#include "hip/hip_runtime.h"
/*
 * @Descripttion: Zhengli's codes
 * @version: 1.0.0
 * @Author: Zhengli
 * @Date: 2024-05-31 12:51:24
 * @LastEditors: Zhengli
 * @LastEditTime: 2024-06-02 20:00:35
 */


#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

//Definition of kernel function to add two variables
//定义了一个核函数，用来进行两个变量的加法运算
__global__ void gpuAdd(int d_a, int d_b, int *d_c) {
	*d_c = d_a + d_b;
}

//main function
//主函数
int main(void) {
	//Defining host variable to store answer
	//定义一个主机上存储结果的变量
	int h_c;
	//Defining device pointer
	//定义一个设备指针
	int *d_c;
	//Allocating memory for device pointer
	//给设备指针分配存储空间
	hipMalloc((void**)&d_c, sizeof(int));
	//Kernel call by passing 1 and 4 as inputs and storing answer in d_c
	//<< <1,1> >> means 1 block is executed with 1 thread per block
	//内核函数被调用后，传递1和4作为输入，同时存储结果到d_c设备比变量中
	//<< <1,1> >>含义是一个块被执行时在每个块中有一个线程
	gpuAdd << <1, 1 >> > (1, 4, d_c);
	//Copy result from device memory to host memory
	//从设备内存中复制结果到主机内存中
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("1 + 4 = %d\n", h_c);
	//Free up memory
	//释放占用的内存空间
	hipFree(d_c);
	return 0;
}
