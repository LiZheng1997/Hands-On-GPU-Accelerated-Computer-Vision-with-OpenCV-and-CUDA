#include "hip/hip_runtime.h"
/*
 * @Descripttion: Zhengli's codes
 * @version: 1.0.0
 * @Author: Zhengli
 * @Date: 2024-05-31 12:51:24
 * @LastEditors: Zhengli
 * @LastEditTime: 2024-06-02 19:55:16
 */

#include <iostream>
#include <stdio.h>
//使用一个__global__函数定义全局的Kernel函数，这个函数是设备上运行的函数，当前定义的是一个空函数。
__global__ void myfirstkernel(void) {
}

int main(void) {
	myfirstkernel << <1, 1 >> >();
	printf("Hello, CUDA!\n");
	return 0;
}
